#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h>

#include <cmath>
#include <cstdlib>
#include <iostream>
#include <random>
#include <cstring>
#include <cassert>


#include "common.hpp"

#define LAMBDA 1.5

//IMPORTANT: ONLY WORKS ON SQUARE MATRICES ATM

__global__ void step(const int *A_outerIndex, const int *A_innerIndex,
                            const double *A_values, const double *b,
                            const unsigned rows, const unsigned cols,
                            const double *sq_norms, double *x, double *X, const unsigned rows_per_thread) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  //printf("Before allocating\n");
  extern __shared__ int data[];
  int *A_outer = data;
  int *A_inner = &data[rows*rows];
  double* A_values_shared = (double*)&A_inner[rows*rows];
  double *X_local = (double*)&A_values_shared[rows*rows];
  //printf("After allocating\n");

  if (tid*rows_per_thread < rows){
    //copy over A to shared memory
    for (unsigned k = 0; k <= rows_per_thread; k++){
      A_outer[tid*rows_per_thread + k] = A_outerIndex[tid*rows_per_thread + k];
    }
    //printf("A_outer\n");
    for (unsigned k = A_outer[tid*rows_per_thread]; k < A_outer[(tid+1)*rows_per_thread]; k++){
      A_inner[k] = A_innerIndex[k];
      A_values_shared[k] = A_values[k];
    }
    //copy over X
    for (unsigned k = A_outer[tid*rows_per_thread]; k < A_outer[(tid+1)*rows_per_thread]; k++){
      X_local[tid*rows + A_inner[k]] = x[A_inner[k]];
      //printf("X at %d: %f\n", A_inner[k], x[A_inner[k]]);
    }
    //printf("A_inner and values\n");
    //perform one update step for assigned row
    for (unsigned local_iter = 0; local_iter < 10; local_iter++){
      for (unsigned k = 0; k < rows_per_thread; k++){
        //printf("Thread: %d, Assigned row: %d\n", tid, tid*rows_per_thread+k);
        // compute dot product row * x
        double dot_product = 0.;
        for (unsigned i = A_outer[tid*rows_per_thread + k]; i < A_outer[tid*rows_per_thread + k + 1]; i++) {
            const double x_value = X_local[tid*rows+A_inner[i]];
            dot_product += A_values_shared[i] * x_value;
        }
        //calculate update
        const double update_coeff = ((b[tid*rows_per_thread + k] - dot_product) / sq_norms[tid*rows_per_thread + k]);
        // save update for x in global matrix, will be used in average step
        for (unsigned i = A_outer[tid*rows_per_thread + k]; i < A_outer[tid*rows_per_thread + k + 1]; i++) {
            const double update = update_coeff * A_values_shared[i];
            X_local[tid*rows + A_inner[i]] += 1.5*update;
            //printf("Update: %f\n", update);
        }
      }
    }

    //set all values back in global matrix for averaging step
    for (int k = 0; k < rows_per_thread; k++){
      for (unsigned i = A_outer[tid*rows_per_thread + k]; i < A_outer[tid*rows_per_thread + k + 1]; i++) {
        X[(tid*rows_per_thread+k)*rows + A_inner[i]] = X_local[tid*rows + A_inner[i]];
      }
    }
  }
}

__global__ void update(const int *A_outerIndex, const int *A_innerIndex,
                            const double *A_values, const double *b,
                            const unsigned rows, const unsigned cols,
                            const double *sq_norms, double *x, double *X, int *affected, const unsigned rows_per_thread) {
  const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid*rows_per_thread < cols){
    for (unsigned k = 0; k < rows_per_thread; k++){
      //sum up updates for assigned entry
      double sum = 0;
      int counter = 0;
      while (true){
          int affecting_thread = affected[(tid*rows_per_thread + k)*rows + counter];
          if (affecting_thread < 0) break;
          counter++;
          const double value = X[affecting_thread*rows + tid*rows_per_thread + k];
          //printf("Update read: %f\n", value);
          sum += value;
      }
      //printf("thread: %d, row: %d, sum: %f, count: %d\n", tid, tid*rows_per_thread + k, sum, counter);
      //if (count > 0.5) printf("total update: %f\n", sum/count);
      //printf("position: %d, x before: %f, ", tid, x[tid]);
      if (counter > 0) x[tid*rows_per_thread + k] = sum/(double)counter;
      //printf("x now: %f\n ", x[tid]);
    }
  }
}


KaczmarzSolverStatus invoke_carp_solver_gpu(
    const int *h_A_outer, const int *h_A_inner, const double *h_A_values,
    const double *h_b, double *h_x, double *h_sq_norms, const unsigned rows,
    const unsigned cols, const unsigned nnz,
    const unsigned max_iterations, const double precision) {



  const unsigned L = 10000;  // we check for convergence every L steps
  bool converged = false;
  assert(rows == cols);

  // allocate move squared norms on device
  double *d_sq_norms;
  hipMalloc((void **)&d_sq_norms, rows * sizeof(double));
  hipMemcpy(d_sq_norms, h_sq_norms, rows * sizeof(double),
             hipMemcpyHostToDevice);

  // move x to device
  double *d_x;
  hipMalloc((void **)&d_x, cols * sizeof(double));
  hipMemcpy(d_x, h_x, cols * sizeof(double), hipMemcpyHostToDevice);


  // move A to device
  int *d_A_outer;
  int *d_A_inner;
  double *d_A_values;
  hipMalloc((void **)&d_A_outer, (rows + 1) * sizeof(int));
  hipMalloc((void **)&d_A_inner, nnz * sizeof(int));
  hipMalloc((void **)&d_A_values, nnz * sizeof(double));
  hipMemcpy(d_A_outer, h_A_outer, (rows + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_A_inner, h_A_inner, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_A_values, h_A_values, nnz * sizeof(double),
             hipMemcpyHostToDevice);
  //std::cout << "Before affecting" << std::endl;
  //calculate indices which affect other rows, and move to device
  std::vector<std::vector<unsigned>> affects(rows); //coding: affects[j][i]: the x at position j is affected by thread i
  for (unsigned k = 0; k < rows; k++){
    for (unsigned i = h_A_outer[k]; i < h_A_outer[k+1]; i++){
      unsigned row = k;
      unsigned col = h_A_inner[i];
      affects.at(row).push_back(col);
    }
  }
  //std::cout << "Affecting middle" << std::endl;
  int* h_affected = new int[rows*rows];
  //std::cout << "Before memsetting" << std::endl;
  std::memset(h_affected, -1, rows*rows*sizeof(int));
  for (int k = 0; k < affects.size(); k++){
    for (int i = 0; i < affects.at(k).size(); i++){
      //std::cout << "K: " << k << " I: " << i << " Outer size: " << affects.size() << " Inner size: " << affects.at(k).size() << std::endl;
      h_affected[k*rows + i] = affects.at(k).at(i);
    }
  }
  //std::cout << "After affecting" << std::endl;
  int *d_affected;
  hipMalloc((void **)&d_affected, rows*rows*sizeof(int));
  hipMemcpy(d_affected, h_affected, rows*rows*sizeof(int), hipMemcpyHostToDevice);


  // move b to device
  double *d_b;
  hipMalloc((void **)&d_b, cols * sizeof(double));
  hipMemcpy(d_b, h_b, cols * sizeof(double), hipMemcpyHostToDevice);

  //move X to device
  double *d_X;
  hipMalloc((void **)&d_X, (cols+1)*cols*sizeof(double));
  hipMemset((void**)d_X, 0, (cols+1)*cols*sizeof(double));

  //calculate nr of blocks and threads
  const int threads_per_block = 512;
  const int blocks = (rows + threads_per_block - 1)/threads_per_block;

  //std::cout << "Blocks: " << blocks << " . Threads per block: " << threads_per_block << std::endl;

  // solve LSE
  for (int iter = 0; iter < max_iterations; iter++){
    step<<<blocks, threads_per_block, 10000>>>(
        d_A_outer, d_A_inner, d_A_values, d_b, rows, cols, d_sq_norms, d_x, d_X, 10);
        auto res = hipDeviceSynchronize();
        assert(res == 0);
    update<<<blocks, threads_per_block>>>(
        d_A_outer, d_A_inner, d_A_values, d_b, rows, cols, d_sq_norms, d_x, d_X, d_affected, 10);
        res = hipDeviceSynchronize();
        assert(res == 0);
    
    //calculate residual every L iterations
    if (iter % 1 == 0 and iter > 0){
      hipMemcpy(h_x, d_x, cols * sizeof(double), hipMemcpyDeviceToHost);
      double residual = 0.0;

      for (unsigned i = 0; i < rows; i++) {
        double dot_product = 0.0;
        for (unsigned j = h_A_outer[i]; j < h_A_outer[i + 1]; j++) {
          dot_product += h_A_values[j] * h_x[h_A_inner[j]];
        }
        residual += (dot_product - h_b[i]) * (dot_product - h_b[i]);
      }
      residual = sqrt(residual);

      printf("Residual: %f\n", residual);

      if (residual < precision) {
        converged = true;
        break;
      }
    }
  }

  // free memory
  hipFree(d_x);
  hipFree(d_X);
  hipFree(d_affected);
  hipFree(d_sq_norms);
  hipFree(d_A_outer);
  hipFree(d_A_inner);
  hipFree(d_A_values);
  hipFree(d_b);

  // check for convergence
  if (converged) return KaczmarzSolverStatus::Converged;
  return KaczmarzSolverStatus::OutOfIterations;
}