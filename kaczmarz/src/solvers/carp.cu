#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h>

#include <cassert>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <random>
#include <set>

#include "common.hpp"
#include "carp_utils.hpp"

#define L_RESIDUAL 100
#define ROWS_PER_THREAD 10
#define LOCAL_RUNS_PER_THREAD 1
#define THREADS_PER_BLOCK 32

// IMPORTANT: ONLY WORKS ON SQUARE MATRICES ATM AND IF ROWS_PER_THREAD DIVIDES
// TOTAL ROWS

KaczmarzSolverStatus invoke_carp_solver_gpu(
    const int *h_A_outer, const int *h_A_inner, const double *h_A_values,
    const double *h_b, double *h_x, double *h_sq_norms, const unsigned rows,
    const unsigned cols, const unsigned nnz, const unsigned max_iterations,
    const double precision, const unsigned max_nnz_in_row, const double b_norm) {
  // check if matrix is square
  assert(rows == cols);
  const unsigned dim = rows;

  // define some variables
  bool converged = false;
  const unsigned total_threads = rows / ROWS_PER_THREAD;

  // allocate move squared norms on device
  double *d_sq_norms;
  hipMalloc((void **)&d_sq_norms, rows * sizeof(double));
  hipMemcpy(d_sq_norms, h_sq_norms, rows * sizeof(double),
             hipMemcpyHostToDevice);

  // move x to device
  double *d_x;
  hipMalloc((void **)&d_x, cols * sizeof(double));
  hipMemcpy(d_x, h_x, cols * sizeof(double), hipMemcpyHostToDevice);

  // move A to device
  int *d_A_outer;
  int *d_A_inner;
  double *d_A_values;
  hipMalloc((void **)&d_A_outer, (rows + 1) * sizeof(int));
  hipMalloc((void **)&d_A_inner, nnz * sizeof(int));
  hipMalloc((void **)&d_A_values, nnz * sizeof(double));
  hipMemcpy(d_A_outer, h_A_outer, (rows + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_A_inner, h_A_inner, nnz * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_A_values, h_A_values, nnz * sizeof(double),
             hipMemcpyHostToDevice);

  // we need to know which values in x are affected by which thread; thats what
  // the below code is for
  std::vector<std::set<unsigned>> affects(
      rows);  // coding: affects[j]: the x at position j is affected by thread
              // in set
  for (unsigned k = 0; k < rows; k++) {
    for (unsigned i = h_A_outer[k]; i < h_A_outer[k + 1]; i++) {
      const unsigned thread = (unsigned)(h_A_inner[i] / ROWS_PER_THREAD);
      affects.at(k).insert(thread);
    }
  }
  const int affects_size = affects.size();

  // move affects to device memory, set to -1 if no thread affects the value
  // (default value)
  int *h_affected = new int[(total_threads + 1) * rows];
  std::memset(h_affected, -1, rows * (total_threads + 1) * sizeof(int));
  // Translate it to a 1D array
  for (int k = 0; k < affects_size; k++) {
    unsigned counter = 0;
    for (const auto &thread : affects[k]) {
      h_affected[k * (total_threads + 1) + counter] = thread;
      counter++;
    }
  }

  // move affects to device
  int *d_affected;
  hipMalloc((void **)&d_affected, rows * (total_threads + 1) * sizeof(int));
  hipMemcpy(d_affected, h_affected, rows * (total_threads + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  // move b to device
  double *d_b;
  hipMalloc((void **)&d_b, cols * sizeof(double));
  hipMemcpy(d_b, h_b, cols * sizeof(double), hipMemcpyHostToDevice);

  //move p, r, q and intermediate storage to device
  double *d_p;
  double *d_r;
  double *d_q;
  double *d_intermediate;
  double *d_zero;
  hipMalloc((void**)&d_p, dim*sizeof(double));
  hipMalloc((void**)&d_r, dim*sizeof(double));
  hipMalloc((void**)&d_q, dim*sizeof(double));
  hipMalloc((void**)&d_intermediate, dim*sizeof(double));
  hipMalloc((void**)&d_zero, dim*sizeof(double));
  hipMemset((void **)d_zero, 0, dim*sizeof(double));

  // move X to device
  double *d_X;
  hipMalloc((void **)&d_X, total_threads * cols * sizeof(double));
  hipMemset((void **)d_X, 0, total_threads * cols * sizeof(double));

  // calculate nr of blocks and threads
  const int blocks =
      (total_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  // solve LSE
  double base_residual = 0;

  //init stuff
  const double relaxation = 1.0;
  dcswp(d_A_outer, d_A_inner,
                     d_A_values, d_b,
                    dim,
                    d_sq_norms, d_x, d_X,
                     relaxation, d_affected, total_threads, d_r, blocks);
  copy_gpu(d_r, d_p, dim);

  for (int iter = 0; iter < max_iterations; iter++) {
    // calculate residual every L_RESIDUAL iterations
    if (iter % L_RESIDUAL == 0) {
      hipMemcpy(h_x, d_x, cols * sizeof(double), hipMemcpyDeviceToHost);
      double residual = 0.0;
      // Calulate residual
      for (unsigned i = 0; i < rows; i++) {
        double dot_product = 0.0;
        for (unsigned j = h_A_outer[i]; j < h_A_outer[i + 1]; j++) {
          dot_product += h_A_values[j] * h_x[h_A_inner[j]];
        }
        residual += (dot_product - h_b[i]) * (dot_product - h_b[i]);
      }
      residual = sqrt(residual);

      // First residual is the base residual
      if (iter == 0) {
        base_residual = residual;
      }

      // debugging output
      printf("Iteration: %d out of %d, Residual/B_norm: %f\n", iter,
             max_iterations, residual/b_norm);

      // check for convergence
      if (residual / base_residual < precision) {
        converged = true;
        break;  // stop all the iterations
      }
    }

    // the real work begins here
    dcswp(d_A_outer, d_A_inner,
                     d_A_values, d_zero,
                    dim,
                    d_sq_norms, d_p, d_X,
                     relaxation, d_affected, total_threads, d_intermediate, blocks);
    add_gpu(d_p, d_intermediate, d_q, -1., dim);
    const double sq_norm_r_old = dot_product_gpu(d_r, d_r, d_intermediate, dim);
    const double alpha = sq_norm_r_old/dot_product_gpu(d_p, d_q, d_intermediate, dim);
    add_gpu(d_x, d_p, d_x, alpha, dim);
    add_gpu(d_r, d_q, d_r, -alpha, dim);
    const double beta = dot_product_gpu(d_r, d_r, d_intermediate, dim)/sq_norm_r_old;
    add_gpu(d_r, d_p, d_p, beta, dim);
  }

  // free memory
  hipFree(d_x);
  hipFree(d_X);
  hipFree(d_affected);
  hipFree(d_sq_norms);
  hipFree(d_A_outer);
  hipFree(d_A_inner);
  hipFree(d_A_values);
  hipFree(d_b);

  // check for convergence
  if (converged) {
    return KaczmarzSolverStatus::Converged;
  } else {
    return KaczmarzSolverStatus::OutOfIterations;
  }
}