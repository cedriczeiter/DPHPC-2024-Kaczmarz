
#include <hip/hip_runtime.h>
#include <cassert>


//write cuda function that takes a matrix and a vector and calculates dot prodcut of every row with the vector added up
__global__ void dot_product_kernel(const double *A, const double *x, double *result, const unsigned rows, const unsigned cols){
    unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < rows){
        double dot_product = 0.0;
        for(unsigned j = 0; j < cols; j++){
            dot_product += A[row * cols + j] * x[j];
        }
        result[row] = dot_product;
    }
}

void dot_product_cuda(const double *h_A, const double *h_x, double *h_result, const unsigned rows, const unsigned cols) {
  double *d_A, *d_x, *d_result;
  hipMalloc(&d_A, rows * cols * sizeof(double));
  hipMalloc(&d_x, cols * sizeof(double));
  hipMalloc(&d_result, rows * sizeof(double));

  hipMemcpy(d_A, h_A, rows * cols * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_x, h_x, cols * sizeof(double), hipMemcpyHostToDevice);

  dot_product_kernel<<<1, rows>>>(d_A, d_x, d_result, rows, cols);

  hipMemcpy(h_result, d_result, rows * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_x);
  hipFree(d_result);
}

//write cuda function that takes a matrix and a vector and calculates squared norm of every row
__global__ void squared_norm_kernel(const double *A, double *result, const unsigned rows, const unsigned cols){
    unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < rows){
        double row_sq_norm = 0.0;
        for(unsigned j = 0; j < cols; j++){
            row_sq_norm += A[row * cols + j] * A[row * cols + j];
        }
        result[row] = row_sq_norm;
    }
}


void squared_norm_cuda(const double *h_A, double *h_result, const unsigned rows, const unsigned cols) {
  double *d_A, *d_result;
  hipMalloc(&d_A, rows * cols * sizeof(double));
  hipMalloc(&d_result, rows * sizeof(double));

  hipMemcpy(d_A, h_A, rows * cols * sizeof(double), hipMemcpyHostToDevice);

  squared_norm_kernel<<<1, rows>>>(d_A, d_result, rows, cols);

  hipMemcpy(h_result, d_result, rows * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_result);
}




/*
KaczmarzSolverStatus dense_kaczmarz(const DenseLinearSystem &lse, double *x,
                                    const unsigned max_iterations,
                                    const double precision,
                                    std::vector<double> &times_residuals,
                                    std::vector<double> &residuals,
                                    std::vector<int> &iterations,
                                    const int convergence_step_rate) {
  const unsigned rows = lse.row_count();
  const unsigned cols = lse.column_count();

  // Calculate the residual norm in the beginning to check for convergence
  double residual_norm_sq = 0.0;
  for (unsigned k = 0; k < rows; k++) {
    double row_residual = 0.0;
    const double *row = lse.A() + k * cols;
    for (unsigned j = 0; j < cols; j++) {
      row_residual += row[j] * x[j];
    }
    row_residual -= lse.b()[k];
    residual_norm_sq += row_residual * row_residual;
  }

  const double residual_norm_0 = std::sqrt(residual_norm_sq);
  const auto start = std::chrono::high_resolution_clock::now();

  double residual_norm_now = 0;  // Preallocate to save allocation overhead

  // Iterate through a maximum of max_iterations
  for (unsigned iter = 0; iter < max_iterations; iter++) {
    // the algorithm has converged iff none of the rows in an iteration caused a
    // substantial correction
    bool substantial_correction = false;

    if (iter % convergence_step_rate == 0) {
      const auto end = std::chrono::high_resolution_clock::now();
      std::chrono::duration<double> elapsed = end - start;
      times_residuals.push_back(elapsed.count());
      double residual_norm_sq = 0.0;
      for (unsigned k = 0; k < rows; k++) {
        double row_residual = 0.0;
        const double *row = lse.A() + k * cols;
        for (unsigned j = 0; j < cols; j++) {
          row_residual += row[j] * x[j];
        }
        row_residual -= lse.b()[k];
        residual_norm_sq += row_residual * row_residual;
      }

      residual_norm_now = std::sqrt(residual_norm_sq);
      residuals.push_back(residual_norm_now /
                          residual_norm_0);  // Takes residual fraction
      iterations.push_back(iter);

      // if residual converged enough, return
      if (residual_norm_now < precision) {
        return KaczmarzSolverStatus::Converged;
      }
    }

    // Process each row of matrix A
    for (unsigned i = 0; i < rows; i++) {
      const double *const a_row = lse.A() + i * cols;
      double dot_product = 0.0;
      double row_sq_norm = 0.0;

      // Compute the dot product and row squared norm
      for (unsigned j = 0; j < cols; j++) {
        dot_product += a_row[j] * x[j];
        row_sq_norm += a_row[j] * a_row[j];
      }

      // Stop if a row squared norm of a row is zero
      if (row_sq_norm < 1e-10) {
        return KaczmarzSolverStatus::ZeroNormRow;
      }

      // Check if the correction is substantial
      const double correction = (lse.b()[i] - dot_product) / row_sq_norm;
      for (unsigned j = 0; j < cols; j++) {
        x[j] += a_row[j] * correction;
      }
    }
  }

  // If it didnt return earlier, then max iterations reached and not
  // converged.
  return KaczmarzSolverStatus::OutOfIterations;
}

KaczmarzSolverStatus sparse_kaczmarz(
    const SparseLinearSystem &lse, Eigen::VectorXd &x,
    const unsigned max_iterations, const double precision,
    std::vector<double> &times_residuals, std::vector<double> &residuals,
    std::vector<int> &iterations, const int convergence_step_rate) {
  const unsigned rows = lse.row_count();
  const unsigned cols = lse.column_count();
  // squared norms of rows of A (so that we don't need to recompute them in
  // each iteration
  Vector sq_norms(rows);
  for (unsigned i = 0; i < rows; i++) {
    sq_norms[i] = lse.A().row(i).dot(lse.A().row(i));
  }

  const double residual_norm_0 = (lse.A() * x - lse.b()).norm();
  const auto start = std::chrono::high_resolution_clock::now();

  double residual_norm_now = 0;  // preallocation

  // same algorithm as in the dense case
  for (unsigned iter = 0; iter < max_iterations; iter++) {
    bool substantial_update = false;

    if (iter % convergence_step_rate == 0) {
      const auto end = std::chrono::high_resolution_clock::now();
      std::chrono::duration<double> elapsed = end - start;
      times_residuals.push_back(elapsed.count());

      residual_norm_now = (lse.A() * x - lse.b()).norm();
      residuals.push_back(residual_norm_now /
                          residual_norm_0);  // Takes residual fraction

      iterations.push_back(iter);

      // if residual small enough, return
      if (residual_norm_now < precision) {
        return KaczmarzSolverStatus::Converged;
      }
    }

    for (unsigned i = 0; i < rows; i++) {
      const auto row = lse.A().row(i);
      const double update_coeff = (lse.b()[i] - row.dot(x)) / sq_norms[i];
      x += update_coeff * row;
    }
  }
  return KaczmarzSolverStatus::OutOfIterations;
}
*/