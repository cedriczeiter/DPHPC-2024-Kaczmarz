#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <stdexcept>
#include "cuda_native.hpp"

KaczmarzSolverStatus native_cuda_solver(const SparseLinearSystem& lse, Vector& x,
                                        const unsigned max_iterations,
                                        const double precision) {
    // Extract matrix data in CSR format
    const auto& A = lse.A();  // Eigen::SparseMatrix
    const auto& b = lse.b();  // Eigen::VectorXd

    const unsigned rows = A.rows();
    const unsigned cols = A.cols();
    const int nnz = A.nonZeros();  // Number of non-zero entries

    // Get CSR pointers from Eigen
    const int* rowPtr = A.outerIndexPtr();  // Row pointers
    const int* colInd = A.innerIndexPtr();  // Column indices
    const double* values = A.valuePtr();    // Non-zero values

    // Allocate device memory
    int *d_rowPtr, *d_colInd;
    double *d_values, *d_b, *d_x;

    hipMalloc((void**)&d_rowPtr, sizeof(int) * (rows + 1));
    hipMalloc((void**)&d_colInd, sizeof(int) * nnz);
    hipMalloc((void**)&d_values, sizeof(double) * nnz);
    hipMalloc((void**)&d_b, sizeof(double) * rows);
    hipMalloc((void**)&d_x, sizeof(double) * cols);

    // Copy data to device
    hipMemcpy(d_rowPtr, rowPtr, sizeof(int) * (rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_colInd, colInd, sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_values, values, sizeof(double) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeof(double) * rows, hipMemcpyHostToDevice);

    // Initialize cuSolverSP handle
    hipsolverSpHandle_t cusolverH = nullptr;
    hipsolverStatus_t status = hipsolverSpCreate(&cusolverH);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        hipFree(d_rowPtr);
        hipFree(d_colInd);
        hipFree(d_values);
        hipFree(d_b);
        hipFree(d_x);
        throw std::runtime_error("Failed to create cuSolverSP handle.");
    }

    // Solve the system using Cholesky factorization
    int singularity = 0;

// Declare and initialize the matrix descriptor
hipsparseMatDescr_t descrA;
hipsparseCreateMatDescr(&descrA);
hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

// Declare cusolver status variable (once)
hipsolverStatus_t status;

// Call the cuSolver function
status = hipsolverSpDcsrlsvchol(
    cusolverH,          // cuSolver handle
    rows,               // Number of rows
    nnz,                // Number of non-zero elements
    descrA,             // Matrix descriptor
    d_values,           // Matrix values (double*)
    d_rowPtr,           // Row pointers (int*)
    d_colInd,           // Column indices (int*)
    d_b,                // Right-hand side vector (double*)
    precision,          // Tolerance
    0,                  // Reorder flag
    d_x,                // Solution vector (double*)
    &singularity        // Singular matrix info
);
    // Check solver status
    if (status != HIPSOLVER_STATUS_SUCCESS || singularity >= 0) {
        hipsolverSpDestroy(cusolverH);
        hipFree(d_rowPtr);
        hipFree(d_colInd);
        hipFree(d_values);
        hipFree(d_b);
        hipFree(d_x);
        return KaczmarzSolverStatus::OutOfIterations;
    }

    // Copy result back to host
    hipMemcpy(x.data(), d_x, sizeof(double) * cols, hipMemcpyDeviceToHost);

    // Free resources
    hipsolverSpDestroy(cusolverH);
    hipFree(d_rowPtr);
    hipFree(d_colInd);
    hipFree(d_values);
    hipFree(d_b);
    hipFree(d_x);

    return KaczmarzSolverStatus::Converged;
}