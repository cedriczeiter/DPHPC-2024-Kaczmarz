#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <stdexcept>
#include "cuda_native.hpp"

KaczmarzSolverStatus native_cuda_solver(const SparseLinearSystem& lse, Vector& x,
                                        const unsigned max_iterations,
                                        const double precision) {
    // Extract matrix data in CSR format
    const auto A_csr = lse.A();  // CSR format matrix
    const auto b = lse.b();
    const unsigned rows = A_csr.rows();
    const unsigned cols = A_csr.cols();
    const int nnz = A_csr.nnz();

    // Allocate device memory
    int *d_A_outerIndex, *d_A_innerIndex;
    double *d_A_values, *d_b, *d_x;

    hipMalloc((void**)&d_A_outerIndex, sizeof(int) * (rows + 1));
    hipMalloc((void**)&d_A_innerIndex, sizeof(int) * nnz);
    hipMalloc((void**)&d_A_values, sizeof(double) * nnz);
    hipMalloc((void**)&d_b, sizeof(double) * rows);
    hipMalloc((void**)&d_x, sizeof(double) * cols);

    // Copy data to device
    hipMemcpy(d_A_outerIndex, A_csr.rowPtr(), sizeof(int) * (rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_A_innerIndex, A_csr.colInd(), sizeof(int) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_A_values, A_csr.values(), sizeof(double) * nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), sizeof(double) * rows, hipMemcpyHostToDevice);

    // Initialize cuSolverSP handle
    hipsolverSpHandle_t cusolverH = nullptr;
    hipsolverStatus_t status = hipsolverSpCreate(&cusolverH);
    if (status != HIPSOLVER_STATUS_SUCCESS) {
        hipFree(d_A_outerIndex);
        hipFree(d_A_innerIndex);
        hipFree(d_A_values);
        hipFree(d_b);
        hipFree(d_x);
        throw std::runtime_error("Failed to create cuSolverSP handle.");
    }

    // Allocate workspace for cuSolverSP
    size_t workspaceSize = 0;
    cusolverSpDcsrlsvchol_bufferSize(cusolverH, rows, nnz, nullptr, d_A_outerIndex, d_A_innerIndex, d_A_values,
                                     nullptr, 0, &workspaceSize);

    void* d_workspace = nullptr;
    hipMalloc(&d_workspace, workspaceSize);

    // Solve the system using Cholesky factorization
    int singularity = 0;
    status = hipsolverSpDcsrlsvchol(cusolverH, rows, nnz, nullptr, d_A_outerIndex, d_A_innerIndex, d_A_values, d_b,
                                   precision, max_iterations, d_x, &singularity);

    // Check solver status
    if (status != HIPSOLVER_STATUS_SUCCESS || singularity >= 0) {
        hipsolverSpDestroy(cusolverH);
        hipFree(d_A_outerIndex);
        hipFree(d_A_innerIndex);
        hipFree(d_A_values);
        hipFree(d_b);
        hipFree(d_x);
        hipFree(d_workspace);
        return KaczmarzSolverStatus::ZeroNormRow;
    }

    // Copy result back to host
    hipMemcpy(x.data(), d_x, sizeof(double) * cols, hipMemcpyDeviceToHost);

    // Free resources
    hipsolverSpDestroy(cusolverH);
    hipFree(d_A_outerIndex);
    hipFree(d_A_innerIndex);
    hipFree(d_A_values);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_workspace);

    return KaczmarzSolverStatus::Converged;
}