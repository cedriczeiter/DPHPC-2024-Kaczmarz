#include "hip/hip_runtime.h"
#include "banded_cuda.hpp"

/**
 * Expects x and A_data padded so that edge cases need not be dealt with.
 */
__global__ void kaczmarz_banded_update(double *x, double *A_data, double *sq_norms, double *b, const int bandwidth) {
  for (unsigned iter = 0; iter < 2'000'000; iter++) {
    for (int row_i = 0; row_i < 2 * bandwidth + 1; row_i++) {
      const int row_idx = threadIdx.x * (2 * bandwidth + 1) + row_i;
      double dot = 0.0;
      for (int i = 0; i < 2 * bandwidth + 1; i++) {
        dot += A_data[(2 * bandwidth + 1) * row_idx + i] * x[row_idx - bandwidth + i];
      }
      const double update_coeff = (b[row_idx] - dot) / sq_norms[row_idx];
      for (int i = 0; i < 2 * bandwidth + 1; i++) {
        x[row_idx - bandwidth + i] += update_coeff * A_data[(2 * bandwidth + 1) * row_idx + i];
      }
      __syncthreads();
    }
  }
}

void invoke_kaczmarz_banded_update(const unsigned bandwidth, const unsigned thread_count, const std::vector<double>& A_data_padded, std::vector<double>& x_padded, const std::vector<double>& sq_norms_padded, const std::vector<double>& b_padded) {
  // copying memory to the GPU
  const auto gpu_malloc_and_copy = [](const std::vector<double>& v) {
    double *gpu_memory;
    const size_t byte_count = v.size() * sizeof(double);
    hipMalloc(&gpu_memory, byte_count);
    hipMemcpy(gpu_memory, &v[0], byte_count, hipMemcpyHostToDevice);
    return gpu_memory;
  };
  double *x_gpu = gpu_malloc_and_copy(x_padded);
  double *A_data_gpu = gpu_malloc_and_copy(A_data_padded);
  double *sq_norms_gpu = gpu_malloc_and_copy(sq_norms_padded);
  double *b_gpu = gpu_malloc_and_copy(b_padded);
  kaczmarz_banded_update<<<1, thread_count>>>(x_gpu + bandwidth, A_data_gpu, sq_norms_gpu, b_gpu, bandwidth);
  hipMemcpy(&x_padded[0], x_gpu, x_padded.size() * sizeof(double), hipMemcpyDeviceToHost);
  hipFree(x_gpu);
  hipFree(A_data_gpu);
  hipFree(sq_norms_gpu);
  hipFree(b_gpu);
}

